#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>

#include <cuda/api_wrappers.hpp>


namespace kernels {

__global__ void from_3D_texture_to_memory_space(hipTextureObject_t texture_source, float* destination, size_t w, size_t h, size_t d) {

	const auto gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	const auto gtidy = threadIdx.y + blockIdx.y * blockDim.y;
	const auto gtidz = threadIdx.z + blockIdx.z * blockDim.z;
	const auto gtid = gtidx + gtidy * w + gtidz * w * h;

	if (gtidx < w && gtidy < h && gtidz < d) {
		destination[gtid] = tex3D<float>(texture_source, gtidx, gtidy, gtidz);
	}
}

__global__ void from_2D_texture_to_memory_space(hipTextureObject_t texture_source, float* destination, size_t w, size_t h) {

	const auto gtidx = threadIdx.x + blockIdx.x * blockDim.x;
	const auto gtidy = threadIdx.y + blockIdx.y * blockDim.y;
	const auto gtid = gtidx + gtidy * static_cast<unsigned>(w);

	if (gtidx < w && gtidy < h) {
		const float x = tex2D<float>(texture_source, gtidx, gtidy);
		printf("thread %u %u, reading value %f, and writing to index %u\n", gtidx, gtidy, x, gtid);
		destination[gtid] = x;
	}
}

}

size_t div_ceil(size_t dividend, size_t divisor) {
	return dividend / divisor + (dividend % divisor == 0 ? 0 : 1);
}

template<class Device>
void array_3d_example(Device& device, size_t w, size_t h, size_t d) {
	const cuda::array::dimensions_t<3> dims = {w, h, d};

	cuda::array::array_t<float, 3> arr(device, dims);
	auto ptr_in = cuda::memory::managed::make_unique<float[]>(arr.size());
	std::iota(ptr_in.get(), ptr_in.get() + arr.size(), 0);
	auto ptr_out = cuda::memory::managed::make_unique<float[]>(arr.size());
	cuda::memory::copy(arr, ptr_in.get());
	cuda::texture_view tv(arr);
	constexpr cuda::grid_block_dimension_t block_dim = 10;
	constexpr cuda::grid_block_dimensions_t block_dims = {block_dim, block_dim, block_dim};
	const cuda::grid_dimensions_t grid_dims = {div_ceil(w, block_dim), div_ceil(h, block_dim), div_ceil(d, block_dim)};
	cuda::launch(kernels::from_3D_texture_to_memory_space, cuda::make_launch_config(grid_dims, block_dims), tv.get(), ptr_out.get(), w, h, d);
	device.synchronize();
	for (size_t i = 0; i < arr.size(); ++i) {
		if (ptr_out[i] != i) {
			std::cout << "ERROR: values were not correctly copied!" << std::endl;
		}
	}

	// copy between arrays and memory spaces
	cuda::array::array_t<float, 3> other_arr(device, dims);
	cuda::memory::copy(other_arr, ptr_out.get());
	cuda::memory::copy(ptr_in.get(), other_arr);

	for (size_t i = 0; i < arr.size(); ++i) {
		if (ptr_in[i] != i) {
			std::cout << "ERROR: values were not correctly copied!" << std::endl;
		}
	}

	// also asynchronously
	auto stream = device.create_stream(cuda::stream::async);
	cuda::memory::async::copy(other_arr, ptr_out.get(), stream);
	cuda::memory::async::copy(ptr_in.get(), other_arr, stream);
	device.synchronize();
	for (size_t i = 0; i < arr.size(); ++i) {
		if (ptr_in[i] != i) {
			std::cout << "ERROR: values were not correctly copied!" << std::endl;
		}
	}
}

template<class Device>
void array_2d_example(Device& device, size_t w, size_t h) {

	const cuda::array::dimensions_t<2> dims = {w, h};
	cuda::array::array_t<float, 2> arr(device , dims);
	auto ptr_in = cuda::memory::managed::make_unique<float[]>(arr.size());
	std::iota(ptr_in.get(), ptr_in.get() + arr.size(), 0);
	auto ptr_out = cuda::memory::managed::make_unique<float[]>(arr.size());

	for (size_t i = 0; i < h; ++i) {
		for (size_t j = 0; j < w; ++j) {
			std::cout << ptr_in[j + i * w] << ' ';
		}
		std::cout << std::endl;
	}

	cuda::memory::copy(arr, ptr_in.get());
	cuda::texture_view tv(arr);

	constexpr cuda::grid_block_dimension_t block_dim = 10;
	constexpr cuda::grid_block_dimensions_t block_dims = {block_dim, block_dim, 1};
	const cuda::grid_dimensions_t grid_dims = {div_ceil(w, block_dim), div_ceil(h, block_dim), 1};

	cuda::launch(kernels::from_2D_texture_to_memory_space, cuda::make_launch_config(grid_dims, block_dims), tv.get(), ptr_out.get(), w, h);
	device.synchronize();
	for (size_t i = 0; i < h; ++i) {
		for (size_t j = 0; j < w; ++j) {
			std::cout << ptr_out[j + i * w] << ' ';
		}
		std::cout << std::endl;
	}
	for (size_t k = 0; k < arr.size(); ++k) {
		if (ptr_out[k] != k) {
			std::cout << "ERROR: values were not correctly copied!" << std::endl;
		}
	}

	// copy between arrays and memory spaces
	cuda::array::array_t<float, 2> other_arr(device, dims);
	cuda::memory::copy(other_arr, ptr_out.get());
	cuda::memory::copy(ptr_in.get(), other_arr);
	for (size_t k = 0; k < arr.size(); ++k) {
		if (ptr_in[k] != k) {
			std::cout << "ERROR: values were not correctly copied!" << std::endl;
		}
	}
	
	// also asynchronously
	auto stream = device.create_stream(cuda::stream::async);
	cuda::memory::async::copy(other_arr, ptr_out.get(), stream);
	cuda::memory::async::copy(ptr_in.get(), other_arr, stream);
	device.synchronize();
	for (size_t k = 0; k < arr.size(); ++k) {
		if (ptr_in[k] != k) {
			std::cout << "ERROR: values were not correctly copied!" << std::endl;
		}
	}
}

int main() {

	auto device = cuda::device::current::get();

	// array dimensions
	constexpr size_t w = 3;
	constexpr size_t h = 3;
	constexpr size_t d = 3;

	array_3d_example(device, w, h, d);
	array_2d_example(device, w, h);
}
