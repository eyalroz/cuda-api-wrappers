#include "hip/hip_runtime.h"
/**
 * An example program utilizing most/all calls from the CUDA
 * Runtime API module:
 *
 *   Stream Management
 *
 */
#include <cuda/runtime_api.hpp>

#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>
#include <thread>

using element_t = float;

using clock_value_t = long long;

__device__ void gpu_sleep(clock_value_t sleep_cycles)
{
    clock_value_t start = clock64();
    clock_value_t cycles_elapsed;
    do { cycles_elapsed = clock64() - start; }
    while (cycles_elapsed < sleep_cycles);
}

template <typename T>
__global__ void add(
	const T* __restrict__  lhs,
	const T* __restrict__  rhs,
	T* __restrict__        result,
	size_t                 length)
{
	auto global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if (global_index < length) {
		result[global_index] = lhs[global_index] + rhs[global_index];
		gpu_sleep(200000);
	}
}

/*
 * Produce a launch configuration with one thread covering each element
 */
cuda::launch_configuration_t make_linear_launch_config(
	const cuda::device_t  device,
	size_t                length)
{
	auto threads_per_block = device.properties().max_threads_per_block();
	cuda::grid::dimension_t num_blocks =
		(length / threads_per_block) +
		(length % threads_per_block == 0 ? 0 : 1);
	return cuda::make_launch_config(num_blocks, threads_per_block, cuda::no_shared_memory);
}

struct buffer_set_t {
	cuda::memory::host::unique_ptr<element_t[]> host_lhs;
	cuda::memory::host::unique_ptr<element_t[]> host_rhs;
	cuda::memory::host::unique_ptr<element_t[]> host_result;
	cuda::memory::device::unique_ptr<element_t[]> device_lhs;
	cuda::memory::device::unique_ptr<element_t[]> device_rhs;
	cuda::memory::device::unique_ptr<element_t[]> device_result;
};

std::vector<buffer_set_t> generate_buffers(
	const cuda::device_t  device,
	size_t                num_kernels,
	size_t                num_elements)
{
	// TODO: This should be an std::array, but generating
	// it is a bit tricky and I don't want to burden the example
	// with template wizardry
	std::vector<buffer_set_t> buffers;
	std::generate_n(std::back_inserter(buffers), num_kernels,
		[&]() {
			return buffer_set_t {
				// Sticking to C++11 here...
				cuda::memory::host::make_unique<element_t[]>(num_elements),
				cuda::memory::host::make_unique<element_t[]>(num_elements),
				cuda::memory::host::make_unique<element_t[]>(num_elements),
				cuda::memory::device::make_unique<element_t[]>(device, num_elements),
				cuda::memory::device::make_unique<element_t[]>(device, num_elements),
				cuda::memory::device::make_unique<element_t[]>(device, num_elements)
			};
		}
	);

	// TODO: Consider actually filling the buffers

	return buffers;
}

int main(int argc, char **argv)
{
	constexpr size_t num_kernels     = 5;
	constexpr size_t num_elements    = 1e7;

	auto device = cuda::device::current::get();
	std::cout << "Using CUDA device " << device.name() << " (having ID " << device.id() << ")\n";

	std::cout << "Generating host buffers... " << std::flush;
	auto buffers = generate_buffers(device, num_kernels, num_elements);
	std::cout << "done.\n" << std::flush;

	std::vector<cuda::stream_t> streams;
	streams.reserve(num_kernels);
	std::generate_n(std::back_inserter(streams), num_kernels,
		[&]() { return device.create_stream(cuda::stream::async); });

	auto common_launch_config = make_linear_launch_config(device, num_elements);
	auto buffer_size = num_elements * sizeof(element_t);

	std::cout
		<< "Running " << num_kernels << " sequences of HtoD-kernel-DtoH, in parallel" << std::endl;
		// Unfortunately, we need to use indices here - unless we
		// had access to a zip iterator (e.g. boost::zip_iterator)
	for(size_t k = 0; k < num_kernels; k++) {
		auto& stream = streams[k];
		auto& buffer_set = buffers[k];
		stream.enqueue.copy(buffer_set.device_lhs.get(), buffer_set.host_lhs.get(), buffer_size);
		stream.enqueue.copy(buffer_set.device_rhs.get(), buffer_set.host_rhs.get(), buffer_size);
		stream.enqueue.kernel_launch(
			add<element_t>,
			common_launch_config,
			buffer_set.device_lhs.get(),
			buffer_set.device_rhs.get(),
			buffer_set.device_result.get(),
			num_elements);
		stream.enqueue.copy(buffer_set.host_result.get(), buffer_set.device_result.get(), buffer_size);
		stream.enqueue.host_function_call(
			[k](cuda::stream_t) {
				std::cout
					<< "Stream " << k+1 << " of " << num_kernels << " has concluded all work. " << std::endl;
			}
		);
	}
	std::this_thread::sleep_for(std::chrono::microseconds(50000));
	for(auto& stream : streams) { stream.synchronize(); }
	cuda::outstanding_error::ensure_none();

	// TODO: Consider checking for correctness here

	std::cout << "\nSUCCESS" << std::endl;
}
