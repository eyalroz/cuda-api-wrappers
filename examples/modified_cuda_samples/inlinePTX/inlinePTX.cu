#include "hip/hip_runtime.h"
/**
 * Derived from the nVIDIA CUDA 8.0 samples by
 *
 *   Eyal Rozenberg <eyalroz@technion.ac.il>
 *
 * The derivation is specifically permitted in the nVIDIA CUDA Samples EULA
 * and the deriver is the owner of this code according to the EULA.
 *
 * Use this reasonably. If you want to discuss licensing formalities, please
 * contact the author.
 */

#include <cuda/runtime_api.hpp>

#include "ptx.cuh"

#include <iostream>
#include <memory>

[[noreturn]] void die_(const std::string& message)
{
	std::cerr << message << "\n";
	exit(EXIT_FAILURE);
}


__global__ void sequence_gpu(int *d_ptr, int length)
{
	int elemID = blockIdx.x * blockDim.x + threadIdx.x;

	if (elemID < length)
	{
		d_ptr[elemID] = ptx::special_registers::laneid();
	}
}

void sequence_cpu(int *h_ptr, int length)
{
	for (int elemID=0; elemID<length; elemID++)
	{
		h_ptr[elemID] = elemID % cuda::warp_size;
	}
}

int main(int argc, char **argv)
{
	if (cuda::device::count() == 0) {
		die_("No CUDA devices on this system");
	}


	const int N = 1000;

	cuda::device::current::set_to_default();
	auto current_device = cuda::device::current::get();

	auto d_ptr = cuda::memory::device::make_unique<int[]>(current_device, N);
	auto h_ptr = cuda::memory::host::make_unique<int[]>(N);

	std::cout << "Generating data on CPU\n";

	sequence_cpu(h_ptr.get(), N);

	cuda::grid::block_dimensions_t cudaBlockSize(256,1,1);
	cuda::grid::dimensions_t cudaGridSize((N + cudaBlockSize.x - 1) / cudaBlockSize.x, 1, 1);
	current_device.launch(
		sequence_gpu,
		{ cudaGridSize, cudaBlockSize },
		d_ptr.get(), N
	);

	cuda::outstanding_error::ensure_none();
	current_device.synchronize();

	auto h_d_ptr = cuda::memory::host::make_unique<int[]>(N);
	cuda::memory::copy(h_d_ptr.get(), d_ptr.get(), N * sizeof(int));

	bool bValid = true;

	for (int i=0; i<N && bValid; i++)
	{
		if (h_ptr.get()[i] != h_d_ptr.get()[i])
		{
			bValid = false;
		}
	}

	std::cout << (bValid ? "SUCCESS" : "FAILURE") << "\n";
	return bValid ? EXIT_SUCCESS: EXIT_FAILURE;
}
