#include "hip/hip_runtime.h"
/**
 * Derived from the nVIDIA CUDA 8.0 samples by
 *
 *   Eyal Rozenberg <eyalroz@technion.ac.il>
 *
 * The derivation is specifically permitted in the nVIDIA CUDA Samples EULA
 * and the deriver is the owner of this code according to the EULA.
 *
 * Use this reasonably. If you want to discuss licensing formalities, please
 * contact the deriving author.
 *
 *
 * This sample illustrates the usage of CUDA streams for overlapping
 * kernel execution with device/host memcopies.  The kernel is used to
 * initialize an array to a specific value, after which the array is
 * copied to the host (CPU) memory.  To increase performance, multiple
 * kernel/memcopy pairs are launched asynchronously, each pair in its
 * own stream.  Devices with Compute Capability 1.1 can overlap a kernel
 * and a memcopy as long as they are issued in different streams.  Kernels
 * are serialized.  Thus, if n pairs are launched, streamed approach
 * can reduce the memcopy cost to the (1/n)th of a single copy of the entire
 * data set.
 *
 * Additionally, this sample uses CUDA events to measure elapsed time for
 * CUDA calls.  Events are a part of CUDA API and provide a system independent
 * way to measure execution times on CUDA devices with approximately 0.5
 * microsecond precision.
 *
 * Elapsed times are averaged over nreps repetitions (10 by default).
 *
 */

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

const char *sSDKsample = "simpleStreams";

const char *sEventSyncMethod[] =
{
	"hipEventDefault",
	"hipEventBlockingSync",
	"hipEventDisableTiming",
	NULL
};

const char *sDeviceSyncMethod[] =
{
	"hipDeviceScheduleAuto",
	"hipDeviceScheduleSpin",
	"hipDeviceScheduleYield",
	"INVALID",
	"hipDeviceScheduleBlockingSync",
	NULL
};

// System includes

// CUDA runtime
#include "hip/hip_runtime.h"

// helper functions and utilities to work with CUDA
#include "../hip/hip_runtime_api.h"

#include <cuda/runtime_api.hpp>

#ifndef WIN32
#include <sys/mman.h> // for mmap() / munmap()
#endif

#include <cstdlib>

#include <fstream>
#include <vector>
#include <iostream>
#include <algorithm>


// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT  4096
#define ALIGN_UP(x,size) ( ((size_t)x+(size-1))&(~(size-1)) )

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i=0; i<num_iterations; i++)
	{
		g_data[idx] += *factor;    // non-coalesced on purpose, to burn time
	}
}

bool correct_data(int *a, const int n, const int c)
{
	for (int i = 0; i < n; i++) {
		if (a[i] != c) {
			std::cout << i << ": " << a[i] << " " << c << "\n";
			return false;
		}
	}
	return true;
}

static const char *sSyncMethod[] =
{
	"0 (Automatic Blocking)",
	"1 (Spin Blocking)",
	"2 (Yield Blocking)",
	"3 (Undefined Blocking Method)",
	"4 (Blocking Sync Event) = low CPU utilization",
	NULL
};

void printHelp()
{
	std::cout
		<< "Usage: " << sSDKsample << " [options below]\n"
		<< "\t--sync_method=n for CPU/GPU synchronization\n"
		<< "\t             n=" << sSyncMethod[0] << "\n"
		<< "\t             n=" << sSyncMethod[1] << "\n"
		<< "\t             n=" << sSyncMethod[2] << "\n"
		<< "\t   <Default> n=" << sSyncMethod[4] << "\n"
		<< "\t--use_generic_memory (default) use generic page-aligned for system memory\n"
		<< "\t--use_cuda_malloc_host (optional) use hipHostMalloc to allocate system memory\n";
}

int main(int argc, char **argv)
{
	int nstreams = 4;               // number of streams for CUDA calls
	int nreps = 10;                 // number of times each experiment is repeated
	int n = 16 * 1024 * 1024;       // number of ints in the data set
	int nbytes = n * sizeof(int);   // number of data bytes
	dim3 threads, blocks;           // kernel launch configuration
	float scale_factor = 1.0f;

	// allocate generic memory and pin it laster instead of using hipHostAlloc()

	int  device_sync_method = hipDeviceScheduleBlockingSync; // by default we use BlockingSync

	int niterations;    // number of iterations for the loop inside the kernel

	if (checkCmdLineFlag(argc, (const char **)argv, "help"))
	{
		printHelp();
		return EXIT_SUCCESS;
	}

	if ((device_sync_method = getCmdLineArgumentInt(argc, (const char **)argv, "sync_method")) >= 0)
	{
		if (device_sync_method == 0 || device_sync_method == 1 || device_sync_method == 2 || device_sync_method == 4)
		{
			std::cout << "Device synchronization method set to = " << sSyncMethod[device_sync_method] << "\n";
			std::cout << "Setting reps to 100 to demonstrate steady state\n";
			nreps = 100;
		}
		else
		{
			std::cout << "Invalid command line option sync_method=\"" << device_sync_method << "\"\n";
			return EXIT_FAILURE;
		}
	}
	else
	{
		printHelp();
		return EXIT_SUCCESS;
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "use_cuda_malloc_host"))
	{
		std::cout
			<< "To simplify this example, support for using cuda_malloc_host instead of "
			<< "pinned memory has been dropped.\n";
		return EXIT_FAILURE;
	}

	std::cout << "\n> ";
	chooseCudaDevice(argc, (const char **)argv);
	auto current_device = cuda::device::current::get();

	// Checking for compute capabilities
	auto properties = current_device.properties();
	auto compute_capability = properties.compute_capability();

	if (compute_capability < cuda::device::compute_capability_t({1, 1}) ) {
		std::cout << properties.name << " does not have Compute Capability 1.1 or newer.  Reducing workload.\n";
	}

	if (compute_capability.major() >= 2) {
		niterations = 5;
	} else {
		if (compute_capability.minor() > 1) {
			niterations = 5;
		} else {
			niterations = 1; // reduced workload for compute capability 1.0 and 1.1
		}
	}

	// Check if GPU can map host memory (Generic Method), if not then we override bPinGenericMemory to be false
	std::cout << "Device: <" << properties.name << "> canMapHostMemory: "
			<< (properties.canMapHostMemory ? "Yes" : "No") << "\n";

	if (not properties.can_map_host_memory())
	{
		std::cout << "Cannot allocate pinned memory (and map GPU device memory to it); aborting.\n";
		return EXIT_FAILURE;
	}

	// Anything that is less than 32 Cores will have scaled down workload
	auto faux_cores_per_sm = compute_capability.max_in_flight_threads_per_processor();
	auto faux_cores_overall = properties.max_in_flight_threads_on_device();
	scale_factor = max((32.0f / faux_cores_overall), 1.0f);
	n = (int)rint((float)n / scale_factor);

	std::cout << "> CUDA Capable: SM " << compute_capability.major() << "." << compute_capability.minor() << " hardware\n";
	std::cout
		<< "> " << properties.multiProcessorCount << " Multiprocessor(s)"
		<< " x " << faux_cores_per_sm << " (Cores/Multiprocessor) = "
		<< faux_cores_overall << " (Cores)\n";

	std::cout << "> scale_factor = " << 1.0f/scale_factor << "\n";
	std::cout << "> array_size   = " << n << "\n\n";

	// enable use of blocking sync, to reduce CPU usage
	std::cout << "> Using CPU/GPU Device Synchronization method " << sDeviceSyncMethod[device_sync_method] << "\n";

	cuda::host_thread_synch_scheduling_policy_t policy;
	switch(device_sync_method) {
	case 0: policy = cuda::heuristic; break;
	case 1: policy = cuda::spin;      break;
	case 2: policy = cuda::yield;     break;
	case 4: policy = cuda::block;     break;
	default: // should not be able to get here
		exit(EXIT_FAILURE);
	}
	current_device.set_synch_scheduling_policy(policy);
	current_device.enable_mapping_host_memory();

	// allocate host memory
	int c = 5;                      // value to which the array will be initialized

	// Allocate Host memory
	auto h_a = cuda::memory::host::make_unique<int[]>(n);

	// allocate device memory
	// pointers to data and init value in the device memory
	auto d_a = cuda::memory::device::make_unique<int[]>(current_device, n);
	auto d_c = cuda::memory::device::make_unique<int>(current_device);
	cuda::memory::copy_single(d_c.get(), &c);

	std::cout << "\nStarting Test\n";

	// allocate and initialize an array of stream handles
	std::vector<cuda::stream_t> streams;
	std::generate_n(
		std::back_inserter(streams), nstreams,
		[&current_device]() {
			// Note: we could omit the specific requirement of synchronization
			// with the default stream, since that's the CUDA default - but I
			// think it's important to state that's the case
			return current_device.create_stream(
				cuda::stream::implicitly_synchronizes_with_default_stream);
		}
	);

	// create CUDA event handles
	// use blocking sync
	auto use_blocking_sync = (device_sync_method == hipDeviceScheduleBlockingSync);

	auto start_event = cuda::event::create(current_device, use_blocking_sync);
	auto stop_event = cuda::event::create(current_device, use_blocking_sync);

	// time memcopy from device
	start_event.record(); // record on the default stream, to ensure that all previous CUDA calls have completed
	cuda::memory::async::copy(h_a.get(), d_a.get(), nbytes, streams[0]);
	stop_event.record();
	stop_event.synchronize(); // block until the event is actually recorded
	auto time_memcpy = cuda::event::time_elapsed_between(start_event, stop_event);
	std::cout << "memcopy:\t" << time_memcpy.count() << "\n";

	// time kernel
	threads=dim3(512, 1);
	blocks=dim3(n / threads.x, 1);
	start_event.record();
	init_array<<<blocks, threads, 0, streams[0].id()>>>(d_a.get(), d_c.get(), niterations);
	stop_event.record();
	stop_event.synchronize();
	auto time_kernel = cuda::event::time_elapsed_between(start_event, stop_event);
	std::cout << "kernel:\t\t" << time_kernel.count() << "\n";

	//////////////////////////////////////////////////////////////////////
	// time non-streamed execution for reference
	threads=dim3(512, 1);
	blocks=dim3(n / threads.x, 1);
	start_event.record();

	for (int k = 0; k < nreps; k++)
	{
		init_array<<<blocks, threads>>>(d_a.get(), d_c.get(), niterations);
		cuda::memory::copy(h_a.get(), d_a.get(), nbytes);
	}

	stop_event.record();
	stop_event.synchronize();
	auto elapsed_time = cuda::event::time_elapsed_between(start_event, stop_event);
	std::cout << "non-streamed:\t" << elapsed_time.count() / nreps << "\n";

	//////////////////////////////////////////////////////////////////////
	// time execution with nstreams streams
	threads=dim3(512,1);
	blocks=dim3(n/(nstreams*threads.x),1);
	memset(h_a.get(), 255, nbytes);     // set host memory bits to all 1s, for testing correctness
	cuda::memory::device::zero(d_a.get(), nbytes); // set device memory to all 0s, for testing correctness
	start_event.record();

	for (int k = 0; k < nreps; k++)
	{
		// asynchronously launch nstreams kernels, each operating on its own portion of data
		for (int i = 0; i < nstreams; i++)
		{
			init_array<<<blocks, threads, 0, streams[i].id()>>>(d_a.get() + i *n / nstreams, d_c.get(), niterations);
		}

		// asynchronously launch nstreams memcopies.  Note that memcopy in stream x will only
		//   commence executing when all previous CUDA calls in stream x have completed
		for (int i = 0; i < nstreams; i++)
		{
			cuda::memory::async::copy(
				h_a.get() + i * n / nstreams,
				d_a.get() + i * n / nstreams, nbytes / nstreams,
				streams[i]);
		}
	}

	stop_event.record();
	stop_event.synchronize();
	elapsed_time = cuda::event::time_elapsed_between(start_event, stop_event);
	std::cout << nstreams <<" streams:\t" << elapsed_time.count() / nreps << "\n";

	// check whether the output is correct
	std::cout << "-------------------------------\n";
	bool bResults = correct_data(h_a.get(), n, c*nreps*niterations);

	std::cout << (bResults ? "SUCCESS" : "FAILURE") << "\n";
	return bResults ? EXIT_SUCCESS : EXIT_FAILURE;
}
