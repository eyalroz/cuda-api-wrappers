#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 * Modifications Copyright (c) 2023, Eyal Rozenberg <eyalroz1@gmx.com>
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes
#include <assert.h>
#include <stdio.h>

#include <climits>
#include <vector>

// CUDA runtime
#include <cuda/api.hpp>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define THREADS_PER_BLOCK 512
#define ALLOWABLE_VARIANCE 1.e-6f
#define NUM_ELEMENTS 8000000

// Stores the square of each input element in output array
__global__ void squareArray(const float *input, float *output,
							int numElements)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numElements) {
		output[idx] = input[idx] * input[idx];
	}
}

// Stores the negative of each input element in output array
__global__ void negateArray(const float *input, float *output,
							int numElements)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numElements) {
		output[idx] = input[idx] * -1;
	}
}

struct negSquareArrays {
	float *input;
	float *square;
	float *negSquare;
	int numElements;
	size_t bytes;
	size_t numBlocks;
};

void fillRandomly(float *array, int numElements)
{
	for (int n = 0; n < numElements; n++) {
		array[n] = rand() / (float) RAND_MAX;
	}
}

void resetOutputArrays(negSquareArrays *hostArrays)
{
	fillRandomly(hostArrays->square, hostArrays->numElements);
	fillRandomly(hostArrays->negSquare, hostArrays->numElements);
}

void prepareHostArrays(negSquareArrays *hostArrays)
{
	hostArrays->numElements = NUM_ELEMENTS;
	size_t bytes = hostArrays->numElements * sizeof(float);

	size_t numBlocks = hostArrays->numElements / (size_t) THREADS_PER_BLOCK;
	if ((numBlocks % (size_t) THREADS_PER_BLOCK) != 0) {
		numBlocks++;
	}

	hostArrays->input = (float *) malloc(bytes);
	hostArrays->square = (float *) malloc(bytes);
	hostArrays->negSquare = (float *) malloc(bytes);
	hostArrays->bytes = bytes;
	hostArrays->numBlocks = numBlocks;

	fillRandomly(hostArrays->input, hostArrays->numElements);
	fillRandomly(hostArrays->square, hostArrays->numElements);
	fillRandomly(hostArrays->negSquare, hostArrays->numElements);
}

cuda::graph::instance_t createFreeGraph(float *dPtr)
{
	hipGraphNode_t freeNode;

	auto graph = cuda::graph::create();
	auto node = graph.insert.node<cuda::graph::node::kind_t::memory_free>(dPtr);
	return graph.instantiate();
}

/**
 * Demonstrates explicitly creating a CUDA graph including memory nodes.
 * createNegateSquaresGraphWithStreamCapture constructs an equivalent graph
 * using stream capture.
 *
 * If d_negSquare_out is non null, then:
 * 1) d_negSquare will not be freed;
 * 2) the value of d_negSquare_out will be set to d_negSquare.
 *
 * Diagram of the graph constructed by createNegateSquaresGraphExplicitly:
 *
 * alloc d_input
 *       |
 * alloc d_square
 *       |
 * Memcpy a to device
 *       |
 * launch kernel squareArray ------->---- Memcpy d_square to host
 *       |                                      |
 * free d_input                                 |
 *       |                                      |
 * allocate d_negSquare                         |
 *       |                                      |
 * launch kernel negateArray -------->--- free d_square
 *       |
 * Memcpy d_negSquare to host
 *       |
 * free d_negSquare
 */
std::pair<cuda::graph::instance_t, float*>
createNegateSquaresGraphExplicitly(int device, negSquareArrays *hostArrays, bool do_neg_squares)
{
	// Array buffers on device
	float *d_input, *d_square, *d_negSquare;

	// Memory allocation parameters
	hipMemAllocNodeParams allocParams;
	memset(&allocParams, 0, sizeof(allocParams));
	allocParams.bytesize = hostArrays->bytes;
	allocParams.poolProps.allocType = hipMemAllocationTypePinned;
	allocParams.poolProps.location.id = device;
	allocParams.poolProps.location.type = hipMemLocationTypeDevice;

	// Kernel launch parameters
	hipKernelNodeParams kernelNodeParams = {0};
	kernelNodeParams.gridDim = dim3(hostArrays->numBlocks, 1, 1);
	kernelNodeParams.blockDim = dim3(THREADS_PER_BLOCK, 1, 1);
	kernelNodeParams.sharedMemBytes = 0;
	kernelNodeParams.extra = NULL;

	hipGraph_t graph;
	hipGraphNode_t allocNodeInput, allocNodeSquare, allocNodeNegSquare;
	hipGraphNode_t copyNodeInput, copyNodeSquare, copyNodeNegSquare;
	hipGraphNode_t squareKernelNode, negateKernelNode;
	hipGraphNode_t freeNodeInput, freeNodeSquare;

	// Buffer for storing graph node dependencies
	std::vector<hipGraphNode_t> nodeDependencies;

	checkCudaErrors(hipGraphCreate(&graph, 0));

	checkCudaErrors(
		hipGraphAddMemAllocNode(&allocNodeInput, graph, NULL, 0, &allocParams));
	d_input = (float *) allocParams.dptr;

	// To keep the graph structure simple (fewer branching dependencies),
	// allocNodeSquare should depend on allocNodeInput
	checkCudaErrors(hipGraphAddMemAllocNode(&allocNodeSquare, graph,
		&allocNodeInput, 1, &allocParams));
	d_square = (float *) allocParams.dptr;

	// copyNodeInput needs to depend on allocNodeInput because copyNodeInput
	// writes to d_input. It does so here indirectly through allocNodeSquare.
	checkCudaErrors(hipGraphAddMemcpyNode1D(
		&copyNodeInput, graph, &allocNodeSquare, 1, d_input, hostArrays->input,
		hostArrays->bytes, hipMemcpyHostToDevice));

	void *squareKernelArgs[3] = {(void *) &d_input, (void *) &d_square,
								 (void *) &(hostArrays->numElements)};
	kernelNodeParams.func = (void *) squareArray;
	kernelNodeParams.kernelParams = (void **) squareKernelArgs;

	// Square kernel depends on copyNodeInput to ensure all data is on the device
	// before kernel launch.
	checkCudaErrors(hipGraphAddKernelNode(&squareKernelNode, graph,
		&copyNodeInput, 1, &kernelNodeParams));

	checkCudaErrors(hipGraphAddMemcpyNode1D(
		&copyNodeSquare, graph, &squareKernelNode, 1, hostArrays->square,
		d_square, hostArrays->bytes, hipMemcpyDeviceToHost));

	// Free of d_input depends on the square kernel to ensure that d_input is not
	// freed while being read by the kernel. It also depends on the alloc of
	// d_input via squareKernelNode > copyNodeInput > allocNodeSquare >
	// allocNodeInput.
	checkCudaErrors(hipGraphAddMemFreeNode(&freeNodeInput, graph,
		&squareKernelNode, 1, d_input));

	// Allocation of C depends on free of A so CUDA can reuse the virtual address.
	checkCudaErrors(hipGraphAddMemAllocNode(&allocNodeNegSquare, graph,
		&freeNodeInput, 1, &allocParams));
	d_negSquare = (float *) allocParams.dptr;

	if (d_negSquare == d_input) {
		printf(
			"Check verified that d_negSquare and d_input share a virtual "
			"address.\n");
	}

	void *negateKernelArgs[3] = {(void *) &d_square, (void *) &d_negSquare,
								 (void *) &(hostArrays->numElements)};
	kernelNodeParams.func = (void *) negateArray;
	kernelNodeParams.kernelParams = (void **) negateKernelArgs;

	checkCudaErrors(hipGraphAddKernelNode(
		&negateKernelNode, graph, &allocNodeNegSquare, 1, &kernelNodeParams));

	nodeDependencies.push_back(copyNodeSquare);
	nodeDependencies.push_back(negateKernelNode);
	checkCudaErrors(hipGraphAddMemFreeNode(&freeNodeSquare, graph,
		nodeDependencies.data(),
		nodeDependencies.size(), d_square));
	nodeDependencies.clear();

	checkCudaErrors(hipGraphAddMemcpyNode1D(
		&copyNodeNegSquare, graph, &negateKernelNode, 1, hostArrays->negSquare,
		d_negSquare, hostArrays->bytes, hipMemcpyDeviceToHost));

	if (d_negSquare_out == NULL) {
		hipGraphNode_t freeNodeNegSquare;
		checkCudaErrors(hipGraphAddMemFreeNode(
			&freeNodeNegSquare, graph, &copyNodeNegSquare, 1, d_negSquare));
	}
	else {
		*d_negSquare_out = d_negSquare;
	}

	checkCudaErrors(hipGraphInstantiate(graphExec, graph, NULL, NULL, 0));
	checkCudaErrors(hipGraphDestroy(graph));
}

cuda::graph::instance_t
createNegateSquaresGraphExplicitly(int device, negSquareArrays *hostArrays)
{
	static constexpr const auto do_neg_squares { true };
	return createNegateSquaresGraphExplicitly(device, hostArrays, do_neg_squares).first;
}

/**
 * Adds work to a CUDA stream which negates the square of values in the input
 * array.
 *
 * If d_negSquare_out is non null, then:
 * 1) d_negSquare will not be freed;
 * 2) the value of d_negSquare_out will be set to d_negSquare.
 *
 * Diagram of the stream operations in doNegateSquaresInStream
 * ---------------------------------------------------------------------
 * | STREAM                             | STREAM2                      |
 * ---------------------------------------------------------------------
 *
 * alloc d_input
 *       |
 * alloc d_square
 *       |
 * Memcpy a to device
 *       |
 * launch kernel squareArray
 *       |
 * record squareKernelCompleteEvent -->-- wait squareKernelCompleteEvent
 *       |                                      |
 * free d_input                                 |
 *       |                                      |
 * allocate d_negSquare                   Memcpy d_square to host
 *       |                                      |
 * launch kernel negateArray                    |
 *       |                                      |
 * record negateKernelCompleteEvent -->-- wait negateKernelCompleteEvent
 *       |                                      |
 * Memcpy d_negSquare to host                   |
 *       |                                free d_square
 * free d_negSquare                             |
 *       |                                      |
 * wait squareFreeEvent --------------<---- record squareFreeEvent
 */
void doNegateSquaresInStream(hipStream_t stream1, negSquareArrays *hostArrays,
							 float **d_negSquare_out = NULL)
{
	float *d_input, *d_square, *d_negSquare;
	hipStream_t stream2;
	hipEvent_t squareKernelCompleteEvent, negateKernelCompleteEvent,
		squareFreeEvent;

	checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));

	checkCudaErrors(hipEventCreate(&squareKernelCompleteEvent));
	checkCudaErrors(hipEventCreate(&negateKernelCompleteEvent));
	checkCudaErrors(hipEventCreate(&squareFreeEvent));

	// Virtual addresses are assigned synchronously when hipMallocAsync is
	// called, thus there is no performace benefit gained by separating the
	// allocations into two streams.
	checkCudaErrors(hipMallocAsync(&d_input, hostArrays->bytes, stream1));
	checkCudaErrors(hipMallocAsync(&d_square, hostArrays->bytes, stream1));

	checkCudaErrors(hipMemcpyAsync(d_input, hostArrays->input, hostArrays->bytes,
		hipMemcpyHostToDevice, stream1));
	squareArray<<<hostArrays->numBlocks, THREADS_PER_BLOCK, 0, stream1>>>(
		d_input, d_square, hostArrays->numElements);
	checkCudaErrors(hipEventRecord(squareKernelCompleteEvent, stream1));

	checkCudaErrors(hipStreamWaitEvent(stream2, squareKernelCompleteEvent, 0));
	checkCudaErrors(hipMemcpyAsync(hostArrays->square, d_square,
		hostArrays->bytes, hipMemcpyDeviceToHost,
		stream2));

	checkCudaErrors(hipFreeAsync(d_input, stream1));
	checkCudaErrors(hipMallocAsync(&d_negSquare, hostArrays->bytes, stream1));
	negateArray<<<hostArrays->numBlocks, THREADS_PER_BLOCK, 0, stream1>>>(
		d_square, d_negSquare, hostArrays->numElements);
	checkCudaErrors(hipEventRecord(negateKernelCompleteEvent, stream1));
	checkCudaErrors(hipMemcpyAsync(hostArrays->negSquare, d_negSquare,
		hostArrays->bytes, hipMemcpyDeviceToHost,
		stream1));
	if (d_negSquare_out == NULL) {
		checkCudaErrors(hipFreeAsync(d_negSquare, stream1));
	}
	else {
		*d_negSquare_out = d_negSquare;
	}

	checkCudaErrors(hipStreamWaitEvent(stream2, negateKernelCompleteEvent, 0));
	checkCudaErrors(hipFreeAsync(d_square, stream2));
	checkCudaErrors(hipEventRecord(squareFreeEvent, stream2));

	checkCudaErrors(hipStreamWaitEvent(stream1, squareFreeEvent, 0));

	checkCudaErrors(hipStreamDestroy(stream2));
	checkCudaErrors(hipEventDestroy(squareKernelCompleteEvent));
	checkCudaErrors(hipEventDestroy(negateKernelCompleteEvent));
	checkCudaErrors(hipEventDestroy(squareFreeEvent));
}

/**
 * Demonstrates creating a CUDA graph including memory nodes using stream
 * capture. createNegateSquaresGraphExplicitly constructs an equivalent graph
 * without stream capture.
 */
cuda::graph::instance_t createNegateSquaresGraphWithStreamCapture(negSquareArrays *hostArrays,
											   float **d_negSquare_out = NULL)
{
	hipGraph_t graph;
	hipStream_t stream;

	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

	checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
	doNegateSquaresInStream(stream, hostArrays, d_negSquare_out);
	checkCudaErrors(hipStreamEndCapture(stream, &graph));

	checkCudaErrors(hipGraphInstantiate(graphExec, graph, NULL, NULL, 0));
	checkCudaErrors(hipStreamDestroy(stream));
	checkCudaErrors(hipGraphDestroy(graph));
}

void prepareRefArrays(negSquareArrays *hostArrays,
					  negSquareArrays *deviceRefArrays,
					  bool **foundValidationFailure)
{
	deviceRefArrays->bytes = hostArrays->bytes;
	deviceRefArrays->numElements = hostArrays->numElements;

	for (int i = 0; i < hostArrays->numElements; i++) {
		hostArrays->square[i] = hostArrays->input[i] * hostArrays->input[i];
		hostArrays->negSquare[i] = hostArrays->square[i] * -1;
	}

	checkCudaErrors(
		hipMalloc((void **) &deviceRefArrays->negSquare, deviceRefArrays->bytes));
	checkCudaErrors(hipMemcpy(deviceRefArrays->negSquare, hostArrays->negSquare,
		hostArrays->bytes, hipMemcpyHostToDevice));

	checkCudaErrors(
		hipMallocManaged((void **) foundValidationFailure, sizeof(bool)));
}

int checkValidationFailure(bool *foundValidationFailure)
{
	if (*foundValidationFailure) {
		std::cout << "Validation FAILURE!\n\n";
		*foundValidationFailure = false;
		return EXIT_FAILURE;
	}
	else {
		std::cout << "Validation PASSED!\n\n";
		return EXIT_SUCCESS;
	}
}

__global__ void validateGPU(float *d_negSquare, negSquareArrays devRefArrays,
							bool *foundValidationFailure)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float ref, diff;

	if (idx < devRefArrays.numElements) {
		ref = devRefArrays.negSquare[idx];
		diff = d_negSquare[idx] - ref;
		diff *= diff;
		ref *= ref;
		if (diff / ref > ALLOWABLE_VARIANCE) {
			*foundValidationFailure = true;
		}
	}
}

void validateHost(negSquareArrays *hostArrays, bool *foundValidationFailure)
{
	float ref, diff;

	for (int i = 0; i < hostArrays->numElements; i++) {
		ref = hostArrays->input[i] * hostArrays->input[i] * -1;
		diff = hostArrays->negSquare[i] - ref;
		diff *= diff;
		ref *= ref;
		if (diff / ref > ALLOWABLE_VARIANCE) {
			*foundValidationFailure = true;
		}
	}
}

int main(int argc, char **argv)
{
	negSquareArrays hostArrays, deviceRefArrays;

	auto launch_config = cuda::launch_config_builder()
		.block_dimensions(THREADS_PER_BLOCK)
		.grid_dimensions(hostArrays.numBlocks)
		.no_dynamic_shared_memory()
		.build();

	// Declare pointers for GPU buffers
	float *d_negSquare = NULL;
	bool *foundValidationFailure = NULL;

	srand(time(0));

	// Being very cavalier about our command-line arguments here...
	cuda::device::id_t device_id = (argc > 1) ? std::stoi(argv[1]) : cuda::device::default_device_id;
	auto device = cuda::device::get(device_id);

	if (cuda::version_numbers::driver() < cuda::version_numbers::make(11040)) {
		std::cout << "Waiving execution as driver does not support Graph Memory Nodes\n";
		exit(EXIT_SUCCESS);
	}

	if (not device.supports_memory_pools()) {
		std::cout << "Waiving execution as device does not support Memory Pools\n";
		exit(EXIT_SUCCESS);
	}
	std::cout << "Setting up sample.\n";

	prepareHostArrays(&hostArrays);
	prepareRefArrays(&hostArrays, &deviceRefArrays, &foundValidationFailure);
	auto stream = device.create_stream(cuda::stream::async);
	std::cout << "Setup complete.\n\n";

	std::cout << "Running negateSquares in a stream.\n";
	doNegateSquaresInStream(stream.handle(), &hostArrays);
	std::cout << "Validating negateSquares in a stream...\n";
	validateHost(&hostArrays, foundValidationFailure);
	checkValidationFailure(foundValidationFailure);
	resetOutputArrays(&hostArrays);

	{
		std::cout << "Running negateSquares in a stream-captured graph.\n";
		auto executable_graph_instance = createNegateSquaresGraphWithStreamCapture(&hostArrays);
		cuda::graph::launch(executable_graph_instance, stream);
		stream.synchronize();
		std::cout << "Validating negateSquares in a stream-captured graph...\n";
		validateHost(&hostArrays, foundValidationFailure);
		checkValidationFailure(foundValidationFailure);
		resetOutputArrays(&hostArrays);
	}

	{
		std::cout << "Running negateSquares in an explicitly constructed graph.\n";
		auto executable_graph_instance = createNegateSquaresGraphExplicitly(device.id(), &hostArrays);
		cuda::graph::launch(executable_graph_instance, stream);
		stream.synchronize();
		std::cout << "Validating negateSquares in an explicitly constructed graph...\n";
		validateHost(&hostArrays, foundValidationFailure);
		checkValidationFailure(foundValidationFailure);
		resetOutputArrays(&hostArrays);
	}
	// Each of the three examples below free d_negSquare outside the graph. As
	// demonstrated by validateGPU, d_negSquare can be accessed by outside the
	// graph before d_negSquare is freed.

	{
		std::cout << "Running negateSquares with d_negSquare freed outside the stream.\n";
		static constexpr const auto compute_neg_squares { true };
		auto pair = createNegateSquaresGraphExplicitly(device.id(), &hostArrays, compute_neg_squares);
		auto executable_graph_instance = std::move(pair.first);
		auto d_negSquare = std::move(pair.second);
		auto free_graph_instance = createFreeGraph(d_negSquare);
		cuda::graph::launch(executable_graph_instance, stream);
		stream.enqueue.kernel_launch(validateGPU, launch_config, d_negSquare, deviceRefArrays, foundValidationFailure);
		stream.synchronize();
		printf(
			"Validating negateSquares with d_negSquare freed outside the "
			"stream...\n");
		validateHost(&hostArrays, foundValidationFailure);
		checkValidationFailure(foundValidationFailure);
		resetOutputArrays(&hostArrays);

		std::cout << "Running negateSquares with d_negSquare freed outside the graph.\n";
		cuda::graph::launch(executable_graph_instance, stream);
		stream.enqueue.kernel_launch(validateGPU, launch_config, d_negSquare, deviceRefArrays, foundValidationFailure);
		stream.synchronize();
		printf(
			"Validating negateSquares with d_negSquare freed outside the graph...\n");
		checkValidationFailure(foundValidationFailure);
		resetOutputArrays(&hostArrays);
		// TODO: What about the instance vs the FreeC?
		printf(
			"Running negateSquares with d_negSquare freed in a different graph.\n");
		cuda::graph::launch(executable_graph_instance, stream);
		stream.enqueue.kernel_launch(validateGPU, launch_config, d_negSquare, deviceRefArrays, foundValidationFailure);
		cuda::graph::launch(free_graph_instance, stream);
		stream.synchronize();
		printf(
			"Validating negateSquares with d_negSquare freed in a different "
			"graph...\n");
		checkValidationFailure(foundValidationFailure);

	}

	std::cout << "\nSUCCESS\n";
}